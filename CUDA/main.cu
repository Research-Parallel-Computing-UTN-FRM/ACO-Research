
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void sum(int *a, int *b, int *c, int N)
{
    int idx = threadIdx.x;
    if (idx < N)
        c[idx] = a[idx] + b[idx];
}

int main(int argc, char *argv[])
{
    int N;
    int *h_a, *h_b, *h_c;
    int *d_a, *d_b, *d_c;

    N = atoi(argv[1]);

    h_a = (int *)malloc(sizeof(int) * N);
    h_b = (int *)malloc(sizeof(int) * N);
    h_c = (int *)malloc(sizeof(int) * N);

    for (int i = 0; i < N - 1; i++)
    {
        h_a[i] = i * 2;
        h_b[i] = i * 3;
    }

    // This pointer to pointer is part of the definition. Dunno why is like that
    hipMalloc(&d_a, sizeof(int) * N);
    hipMalloc(&d_b, sizeof(int) * N);
    hipMalloc(&d_c, sizeof(int) * N);

    hipMemcpy(d_a, h_a, sizeof(int) * N, hipMemcpyKind::hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int) * N, hipMemcpyKind::hipMemcpyHostToDevice);

    printf("HELLO FROM cpu %d\n", N);

    sum<<<1, N + 2>>>(d_a, d_b, d_c, N);

    hipDeviceSynchronize();

    hipMemcpy(h_c, d_c, sizeof(int) * N, hipMemcpyKind::hipMemcpyDeviceToHost);

    printf("a: [");
    for (int i = 0; i < N - 1; i++)
    {
        printf("%d ", h_a[i]);
    }
    printf("]\n");

    printf("b: [");
    for (int i = 0; i < N - 1; i++)
    {
        printf("%d ", h_b[i]);
    }
    printf("]\n");

    printf("c: [");
    for (int i = 0; i < N - 1; i++)
    {
        printf("%d ", h_c[i]);
    }
    printf("]\n");

    free(h_a);
    free(h_b);
    free(h_c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    hipDeviceReset();
    return 0;
}