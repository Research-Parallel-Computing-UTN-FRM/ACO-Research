#include <hiprand/hiprand.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

#define THREADS 10
#define BLOCKS 10
#define N 10
__global__ void kernel_init(hiprandState *state)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(1337, idx, 0, &state[idx]);
}

__global__ void kernel_make_rand(hiprandState *state, float *values)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
        values[idx] = hiprand_uniform(&state[idx]);
}
int main()
{
    float *h_values;
    h_values = (float *)malloc(sizeof(float) * N);

    hiprandState *d_state;
    float *d_values;

    hipMalloc(&d_values, sizeof(float) * N);
    hipMalloc(&d_state, THREADS * BLOCKS);
    kernel_init<<<BLOCKS, THREADS>>>(d_state);

    kernel_make_rand<<<BLOCKS, THREADS>>>(d_state, d_values);

    hipDeviceSynchronize();

    hipMemcpy(h_values, d_values, sizeof(float) * N, hipMemcpyKind::hipMemcpyDeviceToHost);

    printf("a: [");
    for (int i = 0; i < N - 1; i++)
    {
        printf("%f", h_values[i]);
        if (i < N - 2)
            printf(", ");
    }
    printf("]\n");

    free(h_values);

    hipFree(d_values);
    hipFree(d_state);
    hipDeviceReset();

    return 0;
}